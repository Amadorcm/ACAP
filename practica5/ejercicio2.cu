// Ejercicio 2: Coeficiente tanimoto implementando la interseccion en el kernel y el resto en la cpu
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK_ERROR(msg) \
    do { \
        hipError_t err = hipGetLastError(); \
        if (err != hipSuccess) { \
            fprintf(stderr, "[%s] CUDA error: %s: %s. Exiting.\n", \
                msg, hipGetErrorString(err), __FILE__); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

__global__ void intersectionKernel(int *a, int *b, int *inter, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < size) {
        if (a[tid] == 1 && b[tid] == 1) {
            atomicAdd(inter, 1);
        }
    }
}

float calculateTanimoto(int *a, int *b, int size) {
    int inter = 0;

    // Allocate device memory
    int *d_a, *d_b, *d_inter;
    hipMalloc((void **)&d_a, size * sizeof(int));
    hipMalloc((void **)&d_b, size * sizeof(int));
    hipMalloc((void **)&d_inter, sizeof(int));
    CUDA_CHECK_ERROR("CUDA malloc error");

    // Copy data to device
    hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_inter, 0, sizeof(int));
    CUDA_CHECK_ERROR("CUDA memcpy H2D error");

    // Calculate grid size
    int block_size = 512;
    int grid_size = (size + block_size - 1) / block_size;

    // Start timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch kernel
    intersectionKernel<<<grid_size, block_size>>>(d_a, d_b, d_inter, size);
    CUDA_CHECK_ERROR("CUDA kernel launch error");

    // Stop timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copy result back to host
    hipMemcpy(&inter, d_inter, sizeof(int), hipMemcpyDeviceToHost);
    CUDA_CHECK_ERROR("CUDA memcpy D2H error");

    // Free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_inter);

    // Calculate Tanimoto coefficient
    float tanimoto_coefficient = (float)inter / (float)(size - inter + inter);
    printf("Tiempo Tanimoto: %.6f ms\n", milliseconds);

    return tanimoto_coefficient;
}

int main(int argc, char *argv[]) {
    if (argc != 4) {
        printf("Uso: %s <longitud de los vectores> <tamaño del bloque> <semilla>\n", argv[0]);
        exit(EXIT_FAILURE);
    }

    int size = atoi(argv[1]);
    int block_size = atoi(argv[2]);
    int seed = atoi(argv[3]);

    // Allocate host memory
    int *h_a, *h_b;
    h_a = (int *)malloc(size * sizeof(int));
    h_b = (int *)malloc(size * sizeof(int));

    // Initialize vectors randomly
    srand(seed);
    for (int i = 0; i < size; i++) {
        h_a[i] = rand() % 2; // 0 or 1
        h_b[i] = rand() % 2; // 0 or 1
    }

    // Calculate Tanimoto coefficient
    float tanimoto = calculateTanimoto(h_a, h_b, size);

    // Print result
    printf("Tanimoto: %.6f\n", tanimoto);

    // Free memory
    free(h_a);
    free(h_b);

    return 0;
}
